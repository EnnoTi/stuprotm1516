// compile nvcc cusparse_smvp_device.cu -arch=sm_20 -Xcompiler=-fopenmp -lcusparse -o test

// ################################################################################################
//				  Header: cusparse_smvp_device
// ------------------------------------Doxygen-Dokumentation---------------------------------------
///  \file cusparse_smvp_device.cu
///  \brief
///  sparse Matrix (im CSR Format) Vektor Produkt, ausgeführt auf dem Device mit Hilfe von Lib: cuSparse
// ------------------------------------------------------------------------------------------------
// ################################################################################################

#include "helper_functions.hpp"
#include "csr_helper_functions.hpp"
#include <iostream>
#include "hipsparse.h"
#define dim 4

void print(double A[], int  m, int n, int lda)
{
	for(int i(0);i<n;++i)
	{
		for (int j(0);j<m;++j)
		{
			std::cout << A[(j*lda)+i] << "  ";
		}
		std::cout << std::endl;
	}
}
void loeschen(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int *nnzPerRowA, double *A, int *csrRowPtrA, int *csrColIndA, double *csrValA)
{
hipsparseDestroy(handle);
hipsparseDestroyMatDescr(descr); 
hipFree(A);
hipFree(csrValA);
hipFree(csrRowPtrA);
hipFree(csrColIndA);
hipFree(nnzPerRowA);
}

int main()
{
 
  hipsparseStatus_t status;
  const int m_dim_matrix(dim),n_dim_matrix(dim),lda(dim);
  double  h_matrix[m_dim_matrix*n_dim_matrix]= {1,0,2,0,3,4,5,0,0,6,7,0,0,0,8,9}; 
  std::cout << "Die Matrix A_h: " << std::endl;
  print(h_matrix,m_dim_matrix,n_dim_matrix,lda);
  
  
  double *d_matrix;
  int *nnzPerRow; 
  int nonzero = 0;
  
 
  
  hipMalloc(&d_matrix,sizeof(double)*m_dim_matrix*n_dim_matrix);
  hipMalloc(&nnzPerRow,sizeof(int)*n_dim_matrix);
  
  hipMemcpy(d_matrix,&h_matrix,sizeof(double)*n_dim_matrix*m_dim_matrix, hipMemcpyHostToDevice);
  
  
  
  hipsparseHandle_t handle=NULL;
  hipsparseMatDescr_t descr=NULL;
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descr); 
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL); // Beschreibt Allgemeine Matrix (z.B. könnte die Matrix symmetrisch sein und so wir ein anderer algorithmus verwendet)
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);// Setzt ob der Erste Matrix eintrag a11 ist oder a00
  
  status =  hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, m_dim_matrix, n_dim_matrix, descr, d_matrix, lda, nnzPerRow, &nonzero);
 
   if (status != HIPSPARSE_STATUS_SUCCESS)
   {
     std::cout << "Error "<<std::endl;
     loeschen(handle, descr, nnzPerRow, d_matrix, NULL, NULL, NULL);
     return 0;
   }
  std::cout << "Anzahl der nicht null Elemente  " << nonzero << std::endl;
    
    
  double *csrValA;
  int *csrRowPtrA;
  int *csrColIndA;
  
  
  hipMalloc(&csrValA,sizeof(float)*nonzero);
  hipMalloc(&csrRowPtrA,sizeof(int)*(n_dim_matrix+1));
  hipMalloc(&csrColIndA,sizeof(int)*nonzero);
  
status = hipsparseDdense2csr(handle, m_dim_matrix, n_dim_matrix, descr, d_matrix, lda, nnzPerRow, csrValA, csrRowPtrA, csrColIndA);
  if (status != HIPSPARSE_STATUS_SUCCESS)
   {
     std::cout << "Error "<<std::endl;
     loeschen(handle, descr, nnzPerRow, d_matrix, csrRowPtrA, csrColIndA, csrValA);
     return 0;
   }
   
hipDeviceSynchronize();
  
  
    
float alpha(1);
float beta(1);
float h_x[m_dim_matrix]={1,2,3,4};
float h_y[m_dim_matrix];
float *d_x;
float *d_y;



  
hipMalloc(&d_x,sizeof(float)*m_dim_matrix);
hipMalloc(&d_y,sizeof(float)*m_dim_matrix);
hipMemcpy(d_x,&h_x,sizeof(float)*m_dim_matrix, hipMemcpyHostToDevice);
hipMemcpy(d_y,&h_y,sizeof(float)*m_dim_matrix, hipMemcpyHostToDevice);
  
  

 status = hipsparseScsrmv((hipsparseHandle_t)handle,
			 (hipsparseOperation_t)HIPSPARSE_OPERATION_TRANSPOSE,
			 (int)m_dim_matrix,
			 (int)n_dim_matrix,
			 (int)nonzero,
			 (const float *) &alpha,
			 (const hipsparseMatDescr_t)descr,
			 (const float *)csrValA,
			 (const int *)csrRowPtrA,
			 (const int *)csrColIndA,
			 (const float *)d_x,
			 (const float *)&beta,
			 (float *)d_y);

  if (status != HIPSPARSE_STATUS_SUCCESS)
   {
     std::cout << "Error "<<std::endl;
     loeschen(handle, descr, nnzPerRow, d_matrix, csrRowPtrA, csrColIndA, csrValA);
     hipFree(d_y);
     hipFree(d_x);
     return 0;
   }

     
  hipMemcpy(&h_y,d_y,sizeof(float)*m_dim_matrix, hipMemcpyDeviceToHost);
  
std::cout << std::endl;
std::cout << "y: " << std::endl;
//print(X,n,m,ldc);
print_float(h_y,m_dim_matrix);
std::cout << std::endl;


  
  
  
  // Befreie reservierten Speicher, handle, beschreibung der Matrix
  loeschen(handle, descr, nnzPerRow, d_matrix, csrRowPtrA, csrColIndA, csrValA);
    
}
