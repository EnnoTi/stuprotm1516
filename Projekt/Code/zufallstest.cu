#include "hip/hip_runtime.h"
﻿// #################################################################################################
//			Studienprojekt Modellbildung & Simulation - 2015/16
// #################################################################################################
// 					zufallstest.cpp
// ------------------------------------Doxygen-Dokumentation----------------------------------------
///  \file zufallstest.cpp
///  \brief
///  Testet spmv, gpudefect, (spmv2) und vergleicht die ergebnisse mit den Implementationen aus DIA.hpp 
///  
//#################################################################################################

#include"gpumatoperations.hpp"
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <time.h>   

using namespace std;
// testet  spmv, gpudefect, (spmv2) für eine Matrix mit vorgegebener Bandstruktur, vorgegebenem Datentyp und zufälligen Einträgen zwischen -100 und 100 (Einträge sind scheinbar nicht wirklich zufällig) 
template<typename type>
void generatetest(int dim, int ndiags, Vector<int>& offset)
{
    Vector<type> x(dim);
    Vector<type> b(dim);
    Vector<type> res(dim);
    Vector<type> data (dim*ndiags);
    srand (time(NULL));
	//setze x Daten
    for (int i=0; i< dim; ++i){

		//srand (time(NULL));
        x[i]=(static_cast <type> ((rand()) / static_cast <type> (RAND_MAX))*200 - 100);
    }
    //setze Matrix Daten
    for (int i=0; i< ndiags; ++i){
        if(offset[i]<=0){
            for (int j=-offset[i]; j< dim; ++j){
				//srand (time(NULL));
                data[i*dim+j]=((static_cast <type> (rand()) / static_cast <type> (RAND_MAX))*200 - 100);
            }
        }else
        {
            for (int j=0; j< dim-offset[i]; ++j){
				//srand (time(NULL));
                data[i*dim+j]=((static_cast <type> (rand()) / static_cast <type> (RAND_MAX))*200 - 100);
            }
        }
    }
    DIA<type> mat (dim, ndiags, data, offset);
    cout<<"Matrix erstellt"<<endl;
    //cout<<mat.checkIntact()<<endl;

    spmv(b, mat, x);
    defect(res, mat, b, x);
    cout<<"norm gpu ergebnis: "<<norm(b)<<endl;
    cout<<"norm defekt: "<<norm(res)<<endl;
    gpudefect(res, mat, b, x);
    cout<<"norm gpudefekt: "<<norm(res)<<endl;
	
	/*	
	spmv2(b, mat, x);
	defect(res, mat, b, x);
	cout<<"norm gpu2 ergebnis: "<<norm(b)<<endl;
	cout<<"norm cpu defekt: "<<norm(res)<<endl;
    */
    matvec2(b, mat, x);
	gpudefect(res, mat, b, x);
	cout<<"norm cpu ergebnis: "<<norm(b)<<endl;
    cout<<"norm gpudefekt: "<<norm(res)<<endl;
    defect(res, mat, b, x);
    cout<<"norm defekt: "<<norm(res)<<endl;

}
int main (){
    int dim (1000000);
    int ndiags(7);
    Vector<int> offset(ndiags);
    offset[0]=-10000;
    offset[1]=-100;
    offset[2]=-1;
    offset[3]=0;
    offset[4]=1;
    offset[5]=100;
    offset[6]=10000;
    
    generatetest<double>(dim, ndiags, offset);
}

